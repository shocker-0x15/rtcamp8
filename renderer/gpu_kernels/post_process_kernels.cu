#include "hip/hip_runtime.h"
#define PURE_CUDA
#include "../renderer_shared.h"

using namespace rtc8;
using namespace rtc8::shared;
using namespace rtc8::device;



CUDA_DEVICE_FUNCTION CUDA_INLINE RGBSpectrum applySimpleToneMap(
    const RGBSpectrum &input, float brightness) {
    RGBSpectrum color = input;
    float lum = input.luminance();
    if (lum > 0.0f) {
        float lumT = 1 - std::exp(-brightness * lum);
        color *= lumT / lum;
    }
    return color;
}

CUDA_DEVICE_KERNEL void applyToneMap() {
    uint2 launchIndex = make_uint2(blockDim.x * blockIdx.x + threadIdx.x,
                                   blockDim.y * blockIdx.y + threadIdx.y);
    if (launchIndex.x >= plp.s->imageSize.x ||
        launchIndex.y >= plp.s->imageSize.y)
        return;

    RGBSpectrum accumResult = plp.s->accumBuffer.read(launchIndex);
    constexpr float gamma = 1.0f / 0.6f;
    accumResult.r = std::pow(accumResult.r, gamma);
    accumResult.g = std::pow(accumResult.g, gamma);
    accumResult.b = std::pow(accumResult.b, gamma);
    RGBSpectrum output = applySimpleToneMap(accumResult, /*plp.f->brighness*/1.0f);

    //constexpr float gamma = 1.0f / 0.6f;
    //output.r = std::pow(output.r, gamma);
    //output.g = std::pow(output.g, gamma);
    //output.b = std::pow(output.b, gamma);

    plp.f->outputBuffer.write(launchIndex, output);
}
